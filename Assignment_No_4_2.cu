/*
 * Problem Statement :-
    Write a CUDA Program using CUDA C for :
        1. Addition of two large vectors
        2. Matrix Multiplication 
*/

//2. Matrix Multiplication 

#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024
#define THREADS_PER_BLOCK 32


// The blockIdx variable contains the index of the current block
// threadIdx contains the index of the current thread within its block
// The blockDim variable contains the dimensions of the block
// blockIdx.x and blockIdx.y are the x and y indices of the block 
// threadIdx.x and threadIdx.y are the x and y indices of the thread within the block
__global__ void matrix_multiply(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) 
    {
        int sum = 0;
        for (int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main()
{
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    int size = N * N * sizeof(int);

    // Allocate memory on host
    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N * N; i++)
    {
        h_a[i] = 1;
        h_b[i] = 2;
        h_c[i] = 0;
    }

    // Allocate memory on device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy input data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel on device
    dim3 grid((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1);
    dim3 block(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
    matrix_multiply<<<grid, block>>>(d_a, d_b, d_c, N);

    // Copy output data from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; i++) 
    {
        for (int j = 0; j < N; j++) 
        {
            printf("%d ", h_c[i * N + j]); 
        }
        printf("\n");
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free memory on host
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}